#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <unistd.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>


// timing of functions
clock_t start,end;


/* invert and square every element of the input array in parallel */
__global__ void _innergpu_2_sqr(double *psum, int64_t panel)
{
    int64_t globalIdx;
    int64_t tndx = threadIdx.x;
    int64_t bndx=  blockIdx.x;
    int64_t bdim=  blockDim.x;

    globalIdx = bdim * ( bndx + panel * bdim) + tndx + 1;

    double  r = 1.0/(double)globalIdx;
    double r2 = r*r;
    //printf("bdim = %li, bndx = %li, tndx = %li, globalIdx = %li\n",bdim,bndx,tndx,globalIdx);
    atomicAdd(&psum[tndx],r2);
}

int main()
{

    int64_t   N = 16000000000;
    //int64_t   N = 1600000;
    //int Nthr=1024;
    int i , Nthr=1024;
    int64_t Nblocks,Npanels, j;

    double sum = 0.0;
    int64_t _N;

    _N = N - (N %  Nthr);
    Npanels = 1;
    Nblocks = (int64_t)ceil((double)_N/(double)Nthr);
    //printf(" original: N = %li, _N = %li, Npanels = %li, Nblocks = %li\n",N,_N,Npanels,Nblocks);
    if (Nblocks > Nthr*Nthr) {
        Nblocks = Nthr*Nthr;
        Npanels = (int64_t)ceil((double)_N/((double)Nthr*(double)Nblocks));
    }

    double *ps_h, *ps_d;
    start = std::clock();
    //printf("post: N = %li, _N = %li, Npanels = %li, Nblocks = %li\n",N,_N,Npanels,Nblocks);
    ps_h = (double*)calloc(Nthr,sizeof(double));
    hipMalloc(  &ps_d, Nthr * sizeof(double) );
    hipMemcpy(ps_d,ps_h,Nthr*sizeof(double), hipMemcpyHostToDevice);

    // we will handle the remaining portions on CPU, here
    for(i = N ; i>_N; i--) {
      sum += pow(1.0/(double)i,2.0);
    }

    // first compute inverse square
    for (j=0;j<Npanels;j++) {
       _innergpu_2_sqr<<< Nblocks, Nthr >>>(ps_d,j);
    }
    //_innergpu_2_sqr<<< _N, 1 >>>(ps_d);
    hipMemcpy(ps_h,ps_d,Nthr*sizeof(double), hipMemcpyDeviceToHost);
    for(i = 0 ; i<Nthr; i++) {
      //printf("ps_h[%li]=%18.15f, sum=%18.15f\n",i,ps_h[i],sum);
      sum += ps_h[i];
    }

    end = std::clock();


    // Clean up
    hipFree(ps_d);
    free(ps_h);

    printf("sum = %18.16f\n",sum);
}
