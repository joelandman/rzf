#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <unistd.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>


// timing of functions
clock_t start,end;


/* invert and square every element of the input array in parallel */
__global__ void _innergpu_2_sqr(double *psum,int64_t panel)
{
    int64_t globalIdx;
    globalIdx = (panel - 1) * blockDim.x*blockDim.x +
        blockIdx.x * blockDim.x + threadIdx.x + 1;
    int ndx = threadIdx.x;
    double  r = 1.0/(double)globalIdx;
    double r2 = r*r;
    atomicAdd(&psum[ndx],r2);
}

int main()
{

    //int64_t   N = 16000000000;
    int64_t   N = 160000;
    //int Nthr=1024;
    int i , Nthr=1024;
    int64_t Nblocks,Npanels, j;

    double sum = 0.0;
    int64_t _N;

    _N = N - (N %  Nthr);
    Npanels = 1;
    Nblocks = (int64_t)ceil((double)_N/(double)Nthr);
    if (Nblocks > Nthr*Nthr) {
        Nblocks = Nthr*Nthr;
        Npanels = (int64_t)ceil((double)_N/((double)Nthr*(double)Nblocks));
    }

    double *ps_h, *ps_d;
    start = std::clock();

    ps_h = (double*)calloc(Nthr,sizeof(double));
    hipMalloc(  &ps_d, Nthr * sizeof(double) );
    hipMemcpy(ps_d,ps_h,Nthr*sizeof(double), hipMemcpyHostToDevice);

    // we will handle the remaining portions on CPU, here
    for(i = N ; i>_N; i--) {
      sum += pow(1.0/(double)i,2.0);
    }
    printf("Npanels=%lli, Nblocks=%lli\n",Npanels,Nblocks);

    // first compute inverse square
    for (j=0;j<Npanels;j++) {
       _innergpu_2_sqr<<< Nblocks, Nthr >>>(ps_d,j);
    }
    //_innergpu_2_sqr<<< _N, 1 >>>(ps_d);
    hipMemcpy(ps_h,ps_d,Nthr*sizeof(double), hipMemcpyDeviceToHost);
    for(i = 0 ; i<Nthr; i++) {
      //printf("ps_h[%lld]=%18.15f, sum=%18.15f\n",i,ps_h[i],sum);
      sum += ps_h[i];
    }

    end = std::clock();


    // Clean up
    hipFree(ps_d);
    free(ps_h);

    printf("sum = %18.16f\n",sum);
}
