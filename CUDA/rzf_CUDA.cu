#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <unistd.h>
#include <stdlib.h>
#include <hipcub/hipcub.hpp>

#include <hip/hip_runtime.h>
#include <cstring>
#include <ctime>

#include "Utilities.cuh"

#include <iostream>

#define M 1024
#define BLOCKSIZE   32
#define WARP_SIZE 32

// timing of functions
clock_t start,end;

/* invert and square every element of the input array in parallel */
__global__ void
_cuda_create_inverted_squared_array(double *in, unsigned int n, unsigned int offset)
{
    unsigned int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    while(globalIdx < n)
    {
        in[globalIdx] = 1.0/(__uint2double_rn(globalIdx+offset) * __uint2double_rn(globalIdx+offset));
        globalIdx += blockDim.x * gridDim.x;
    }
}

// Riemann zeta function (2)
void cuda_riemann_zeta_function(int N) {
    // Get device properties to compute optimal launch bounds
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int pagesize=getpagesize();
    double sum = 0.0, *h_result = (double *)malloc((M / BLOCKSIZE) * sizeof(double))
    int _N;
    int num_SMs = prop.multiProcessorCount;

    // cheat a little ... reduce N by N mod (num_SMs*1024), so that
    // we don't have to pad with zeros, or put conditionals in
    // the vector flow
    _N = N - (N % (num_SMs * 1024));

    // we will handle the remaining portions on CPU, here
    for(i = N ; i>_N; i--) {
       sum += pow(1.0/(double)i,2.0);
    }


    double * d_a;
    start = std::clock();

    // create array on GPU
    hipMalloc( (void**) &d_a, _N * sizeof(double) );


    // first compute inverse square
    _cuda_create_inverted_squared_array<<< num_SMs, 1024 >>>(d_a, scalar, _N);

    // second, parallel sum reduction over d_a
    reduce6();
    end = std::clock();


    // Clean up
    hipFree(d_a);
    free(a);
}

/**************************/
/* BLOCK REDUCTION KERNEL */
/**************************/
__global__ void sum(const double * __restrict__ indata, double * __restrict__ outdata) {

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // --- Specialize BlockReduce for type float.
    typedef hipcub::BlockReduce<double, BLOCKSIZE> BlockReduceT;

    // --- Allocate temporary storage in shared memory
    __shared__ typename BlockReduceT::TempStorage temp_storage;

    float result;
    if(tid < N) result = BlockReduceT(temp_storage).Sum(indata[tid]);

    // --- Update block reduction value
    if(threadIdx.x == 0) outdata[blockIdx.x] = result;

    return;
}



int main()
{
 double  y=0.0;
 int i,N = 100000000;

  for(i=N;i>=1;i--) {
     y += pow(1.0/(double)i,2.0);
  }

 printf("[index decreasing] sum = %18.16f\n",y);
}
